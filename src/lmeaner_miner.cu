#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018 Jiri Vadura - photon
// This CUDA part of Theta optimized miner is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include <sys/time.h> // gettimeofday
#include "cuckoo.h"
#include "siphash.cuh"
#include "blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;

typedef u32 node_t;
typedef u64 nonce_t;

#ifndef XBITS
#define XBITS 6
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES ((node_t)1 << NODEBITS)
#define NODEMASK (NNODES - 1)

#define YBITS XBITS
#define ZBITS (EDGEBITS - XBITS - YBITS)
const static u32 NX        = 1 << XBITS;
const static u32 NX2       = NX * NX;
const static u32 XMASK     = NX - 1;
const static u32 X2MASK    = NX2 - 1;
const static u32 NY        = 1 << YBITS;
const static u32 NZ        = 1 << ZBITS;

#define DUCK_SIZE_A 130LL
#define DUCK_SIZE_B 83LL

#define DUCK_A_EDGES (DUCK_SIZE_A * 1024LL)
#define DUCK_A_EDGES_64 (DUCK_A_EDGES * 64LL)

#define DUCK_B_EDGES (DUCK_SIZE_B * 1024LL)
#define DUCK_B_EDGES_64 (DUCK_B_EDGES * 64LL)

#ifndef CTHREADS
#define CTHREADS 1024
#endif

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ __forceinline__ ulonglong4 Pack8(const u32 e0, const u32 e1, const u32 e2, const u32 e3, const u32 e4, const u32 e5, const u32 e6, const u32 e7) {
  return make_ulonglong4((u64)e0<<32|e1, (u64)e2<<32|e3, (u64)e4<<32|e5, (u64)e6<<32|e7);
}

__global__  void FluffySeed2A(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, int * __restrict__ indexes) {
  const int col = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHA = 32;
  const int FLUSHA2 = 64;

  __shared__ u16 tmp[NX][FLUSHA2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(tmp[0][0]);
  __shared__ int counters[NX];

  for (int r = lid; r < NX; r += dim)
    counters[r] = 0;
  __syncthreads();
  const int loops = (NEDGES / NX) / dim;

  for (int i = 0; i < loops; i++) {
    u64 nonce = col * (NEDGES / NX) + i * dim + lid;
    u32 hashx = dipnode(sipkeys, nonce, 0);
    int row = hashx & XMASK;
    int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1));
    tmp[row][counter] = (u16)nonce;
    __syncthreads();
    if (counter == FLUSHA-1) {
      int localIdx = min(FLUSHA2, counters[row]);
      int newCount = localIdx - FLUSHA;
      counters[row] = newCount;
      int cnt = min((int)atomicAdd(indexes + row * NX + col, FLUSHA), (int)(DUCK_A_EDGES - FLUSHA));
      for (int i = 0; i < FLUSHA; i += TMPPERLL4)
        buffer[((u64)(row * NX + col) * DUCK_A_EDGES + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[row][t] = tmp[row][t + FLUSHA];
      }
    }
  }
  __syncthreads();
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = 0;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + row * NX + col, TMPPERLL4), (int)(DUCK_A_EDGES - TMPPERLL4));
      buffer[((u64)(row * NX + col) * DUCK_A_EDGES + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

__global__  void FluffySeed2B(const siphash_keys &sipkeys, const u16 * __restrict__ source, ulonglong4 * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB = 32;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ uint2 tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(tmp[0][0]);
  __shared__ int counters[NX];

  for (int r = lid; r < NX; r += dim)
    counters[r] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)sourceIndexes[group], (int)(DUCK_A_EDGES));
  const int loops = (bucketEdges + dim-1) / dim;
  const u64 lag = 16384;
  u64 nonce = (group % NX) * (NEDGES / NX);
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * DUCK_A_EDGES + edgeIndex;
      u16 nonce16 = __ldg(&source[index]);
      if (loop < (loops * 15/16) || nonce16 != 0) { // fillers only at end
        nonce += (((u64)nonce16 - nonce + lag) & 0xffffULL) - lag;
        uint2 edge;
        edge.x = dipnode(sipkeys, nonce, 0);
        edge.y = dipnode(sipkeys, nonce, 1);
        col = (edge.x >> XBITS) & XMASK;
        counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1));
        tmp[col][counter] = edge;
      }
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx - FLUSHB;
      int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, FLUSHB), (int)(DUCK_A_EDGES - FLUSHB));
      for (int i = 0; i < FLUSHB; i += TMPPERLL4)
        destination[((u64)(row * NX + col) * DUCK_A_EDGES + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + FLUSHB];
      }
      counters[col] = newCount;
      assert (newCount < FLUSHB);
    }
    __syncthreads(); 
  }
  if (((nonce - (group % NX) * (NEDGES / NX)) >> 16) != ((NEDGES-1)/NX)>>16) {
    printf("group %x lid %x nonce %08llx\n", group, lid, nonce);
    assert(0);
  }
  
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = e0;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, TMPPERLL4), (int)(DUCK_A_EDGES - TMPPERLL4));
      destination[((u64)(row * NX + col) * DUCK_A_EDGES + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void Increase2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  u32 old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + 4096, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  return (ecounters[word + 4096] & mask) != 0;
}

template<int bktInSize, int bktOutSize>
__global__  void FluffyRound(const int round, const uint2 * __restrict__ source, uint2 * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int lid = threadIdx.x;
  const static int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

  __shared__ u32 ecounters[COUNTERWORDS];

  const int edgesInBucket = min(sourceIndexes[group], bktInSize);
  const int loops = (edgesInBucket + CTHREADS-1) / CTHREADS;

  for (int i = 0; i < COUNTERWORDS / CTHREADS; i++)
    ecounters[lid + CTHREADS * i] = 0; // IS SINGLE-INCREMENT FASTER?
  __syncthreads();
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * CTHREADS + lid;
    if (lindex < edgesInBucket) {
      const int index = bktInSize * group + lindex;
      uint2 edge = __ldg(&source[index]);
      if (edge.x == 0 && edge.y == 0) continue;
      Increase2bCounter(ecounters, (edge.x & EDGEMASK) >> 12);
    }
  }
  __syncthreads();
  for (int loop = loops-1; loop >= 0; loop--) {
    const int lindex = loop * CTHREADS + lid;
    if (lindex < edgesInBucket) {
      const int index = bktInSize * group + lindex;
      uint2 edge = __ldg(&source[index]);
      if (edge.x == 0 && edge.y == 0) continue;
      if (Read2bCounter(ecounters, (edge.x & EDGEMASK) >> 12)) {
        const int bucket = edge.y & X2MASK;
        const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
        destination[bucket * bktOutSize + bktIdx] = make_uint2(edge.y, edge.x);
      }
    }
  }
}

__global__ void FluffyTail(const uint2 *source, uint2 *destination, const int *sourceIndexes, int *destinationIndexes) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  int myEdges = sourceIndexes[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(destinationIndexes, myEdges);
  __syncthreads();
  if (lid < myEdges)
    destination[destIdx + lid] = source[group * DUCK_B_EDGES/4 + lid];
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__global__  void FluffyRecovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];
  
  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < loops; i++) {
    u64 nonce = gid * loops + i;
    u64 u = dipnode(sipkeys, nonce, 0);
    u64 v = dipnode(sipkeys, nonce, 1);
    for (int i = 0; i < PROOFSIZE; i++) {
      if (recoveredges[i].x == u && recoveredges[i].y == v)
        nonces[i] = nonce;
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

struct trimparams {
  u16 ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;
  u16 reportcount;
  u16 reportrounds;

  trimparams() {
    ntrims              =  176;
    genA.blocks         =   NX;
    genA.tpb            = 2*NX;
    genB.blocks         =  NX2;
    genB.tpb            = 2*NX;
    trim.blocks         =  NX2;
    trim.tpb            = CTHREADS;
    tail.blocks         =  NX2;
    tail.tpb            = 1024; // needs to exceed #FINAL EDGES / NX2
    recover.blocks      = 1024;
    recover.tpb         = 1024;
    reportcount         =    1;
    reportrounds        =    0;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  const size_t bufferSize  = DUCK_B_EDGES_64 * NX * sizeof(uint2);
  const size_t bufferSize2 = DUCK_A_EDGES_64 * NX * sizeof(uint2);
  const size_t indexesSize = NX * NY * sizeof(u32);
  int *bufferA;
  int *bufferB;
  int *indexesE;
  int *indexesE2;
  u32 hostA[NX * NY];
  u32 *uvnodes;
  proof sol;
  siphash_keys sipkeys, *dipkeys;

  edgetrimmer(const trimparams _tp) {
    tp = _tp;
    checkCudaErrors(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));

    checkCudaErrors(hipMalloc((void**)&bufferA, bufferSize));
    checkCudaErrors(hipMalloc((void**)&bufferB, bufferSize2));
    checkCudaErrors(hipMalloc((void**)&indexesE, indexesSize));
    checkCudaErrors(hipMalloc((void**)&indexesE2, indexesSize));
  }
  u64 sharedbytes() const {
    return bufferSize + bufferSize2 + 2 * indexesSize;
  }
  ~edgetrimmer() {
    hipFree(bufferA);
    hipFree(bufferB);
    hipFree(indexesE);
    hipFree(indexesE2);
    hipDeviceReset();
  }
  u32 trim() {
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEvent_t startall, stopall;
    checkCudaErrors(hipEventCreate(&startall)); checkCudaErrors(hipEventCreate(&stopall));
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
  
    hipMemset(indexesE, 0, indexesSize);
    hipMemset(indexesE2, 0, indexesSize);
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);
  
    FluffySeed2A<<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4 *)bufferA, (int *)indexesE);
  
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    hipEventRecord(start, NULL);
  
    FluffySeed2B<<<tp.genB.blocks, tp.genB.tpb>>>(*dipkeys, (const u16 *)bufferA, (ulonglong4 *)bufferB, (const int *)indexesE, (int *)indexesE2);
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    printf("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);
  
    hipMemset(indexesE, 0, indexesSize);
    FluffyRound<DUCK_A_EDGES, DUCK_B_EDGES><<<tp.trim.blocks, tp.trim.tpb>>>(0, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
    hipMemset(indexesE2, 0, indexesSize);
    FluffyRound<DUCK_B_EDGES, DUCK_A_EDGES/2><<<tp.trim.blocks, tp.trim.tpb>>>(1 ,(const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);
    hipMemset(indexesE, 0, indexesSize);
    FluffyRound<DUCK_A_EDGES/2, DUCK_A_EDGES/4><<<tp.trim.blocks, tp.trim.tpb>>>(2 ,(const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
    hipMemset(indexesE2, 0, indexesSize);
    FluffyRound<DUCK_A_EDGES/4, DUCK_B_EDGES/4><<<tp.trim.blocks, tp.trim.tpb>>>(3 ,(const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);
  
    hipDeviceSynchronize();
  
    for (int round = 4; round < tp.ntrims; round += 2) {
      hipMemset(indexesE, 0, indexesSize);
      FluffyRound<DUCK_B_EDGES/4, DUCK_B_EDGES/4><<<tp.trim.blocks, tp.trim.tpb>>>(round, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
      hipMemset(indexesE2, 0, indexesSize);
      FluffyRound<DUCK_B_EDGES/4, DUCK_B_EDGES/4><<<tp.trim.blocks, tp.trim.tpb>>>(round, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);
    }
    
    hipMemset(indexesE, 0, indexesSize);
    hipDeviceSynchronize();
  
    FluffyTail<<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
    hipMemcpy(hostA, indexesE, NX * NY * sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return hostA[0];
  }
};

#define IDXSHIFT 10
#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = new u64[CUCKOO_SIZE];
  }
  ~cuckoo_hash() {
    delete[] cuckoo;
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

const static u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

const static u32 MAXEDGES = 0x20000;

struct solver_ctx {
  edgetrimmer *trimmer;
  uint2 *edges;
  cuckoo_hash *cuckoo;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices
  u32 us[MAXPATHLEN];
  u32 vs[MAXPATHLEN];

  solver_ctx(const trimparams tp) {
    trimmer = new edgetrimmer(tp);
    edges   = new uint2[MAXEDGES];
    cuckoo  = new cuckoo_hash();
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer->sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete cuckoo;
    delete[] edges;
    delete trimmer;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    soledges[i].x = u2/2;
    soledges[i].y = v2/2;
  }

  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
    recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    assert(ni == PROOFSIZE);
    sols.resize(sols.size() + PROOFSIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
    hipMemset(trimmer->indexesE2, 0, trimmer->indexesSize);
    FluffyRecovery<<<trimmer->tp.recover.blocks, trimmer->tp.recover.tpb>>>(*trimmer->dipkeys, (ulonglong4 *)trimmer->bufferA, (int *)trimmer->indexesE2);
    hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer->indexesE2, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
    checkCudaErrors(hipDeviceSynchronize());
    qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  u32 path(u32 u, u32 *us) {
    u32 nu, u0 = u;
    for (nu = 0; u; u = (*cuckoo)[u]) {
      if (nu >= MAXPATHLEN) {
        while (nu-- && us[nu] != u) ;
        if (~nu) {
          printf("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
          exit(0);
        }
        printf("maximum path length exceeded\n");
        return 0; // happens once in a million runs or so; signal trouble
      }
      us[nu++] = u;
    }
    return nu;
  }

  void addedge(uint2 edge) {
    const u32 u0 = edge.x << 1, v0 = (edge.y << 1) | 1;
    if (u0) {
      u32 nu = path(u0, us), nv = path(v0, vs);
      if (!nu-- || !nv--)
        return; // drop edge causing trouble
      // printf("vx %02x ux %02x e %08x uxyz %06x vxyz %06x u0 %x v0 %x nu %d nv %d\n", vx, ux, e, uxyz, vxyz, u0, v0, nu, nv);
      if (us[nu] == vs[nv]) {
        const u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        const u32 len = nu + nv + 1;
        printf("%4d-cycle found\n", len);
        if (len == PROOFSIZE)
          solution(us, nu, vs, nv);
        // if (len == 2) printf("edge %x %x\n", edge.x, edge.y);
      } else if (nu < nv) {
        while (nu--)
          cuckoo->set(us[nu+1], us[nu]);
        cuckoo->set(u0, v0);
      } else {
        while (nv--)
          cuckoo->set(vs[nv+1], vs[nv]);
        cuckoo->set(v0, u0);
      }
    }
  }

  void findcycles(uint2 *edges, u32 nedges) {
    memset(cuckoo->cuckoo, 0, CUCKOO_SIZE * sizeof(u64));
    for (u32 i = 0; i < nedges; i++)
      addedge(edges[i]);
  }

  int solve() {
    u32 timems,timems2;
    struct timeval time0, time1;

    gettimeofday(&time0, 0);
    u32 nedges = trimmer->trim();
    assert(nedges <= MAXEDGES);
    hipMemcpy(edges, trimmer->bufferA, nedges * 8, hipMemcpyDeviceToHost);
    gettimeofday(&time1, 0);
    timems = (time1.tv_sec-time0.tv_sec)*1000 + (time1.tv_usec-time0.tv_usec)/1000;
    gettimeofday(&time0, 0);
    findcycles(edges, nedges);
    gettimeofday(&time1, 0);
    timems2 = (time1.tv_sec-time0.tv_sec)*1000 + (time1.tv_usec-time0.tv_usec)/1000;
    printf("findcycles edges %d time %d ms total %d ms\n", nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "sb:c:d:h:k:m:n:r:V:v:w:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        printf("SYNOPSIS\n  cuda30 [-d device] [-h hexheader] [-k rounds [-c count]] [-m trims] [-n nonce] [-r range] [-V blocks] [-y threads] [-Z blocks] [-z threads]\n");
        printf("DEFAULTS\n  cuda30 -d %d -h \"\" -k %d -c %d -m %d -n %d -r %d -v %d -w %d -y %d -Z %d -z %d\n", device, tp.reportrounds, tp.reportcount, tp.ntrims, nonce, range, tp.genA.tpb, tp.genB.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'd':
        device = atoi(optarg);
        break;
      case 'k':
        tp.reportrounds = atoi(optarg);
        break;
      case 'c':
        tp.reportcount = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        tp.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'V':
        tp.genA.blocks = atoi(optarg);
        break;
      case 'v':
        tp.genA.tpb = atoi(optarg);
        break;
      case 'w':
        tp.genB.tpb = atoi(optarg);
        break;
      case 'y':
        tp.tail.tpb = atoi(optarg);
        break;
      case 'Z':
        tp.recover.blocks = atoi(optarg);
        break;
      case 'z':
        tp.recover.tpb = atoi(optarg);
        break;
    }
  }
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
  assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;
  printf("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  hipSetDevice(device);

  printf("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, NX);

  solver_ctx ctx(tp);

  // loop starts here
  // wait for header hashes, nonce+r
  u32 sumnsols = 0;
  for (int r = 0; r < range; r++) {
    ctx.setheadernonce(header, sizeof(header), nonce + r);
    printf("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx.trimmer->sipkeys.k0, ctx.trimmer->sipkeys.k1, ctx.trimmer->sipkeys.k2, ctx.trimmer->sipkeys.k3);
    u32 nsols = ctx.solve();
    for (unsigned s = 0; s < nsols; s++) {
      printf("Solution");
      u32* prf = &ctx.sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        printf(" %jx", (uintmax_t)prf[i]);
      printf("\n");
      int pow_rc = verify(prf, &ctx.trimmer->sipkeys);
      if (pow_rc == POW_OK) {
        printf("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          printf("%02x", cyclehash[i]);
        printf("\n");
      } else {
        printf("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
  }
  printf("%d total solutions\n", sumnsols);
  return 0;
}
